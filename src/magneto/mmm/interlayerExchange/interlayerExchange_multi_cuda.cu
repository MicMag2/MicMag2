#include "hip/hip_runtime.h"
/*
 * Copyright 2012, 2013 by the Micromagnum authors.
 *
 * This file is part of MicroMagnum.
 * 
 * MicroMagnum is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * MicroMagnum is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with MicroMagnum.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "config.h"
#include "interlayerExchange_multi_cuda.h"

#include <hip/hip_runtime.h>

#include "config.h"
#include "mmm/constants.h"
#include <iostream>

static const int BLOCK_3D_SIZE_X = 8;
static const int BLOCK_3D_SIZE_Y = 8;
static const int BLOCK_3D_SIZE_Z = 8;

///////////////////////////////////////////////////////////////////
// KERNEL FOR and 3D meshes                                  //
///////////////////////////////////////////////////////////////////

template <typename real, bool periodic_x, bool periodic_y, bool periodic_z>
__global__ 
void kernel_interlayerExchange_multi_3d(
	const real *Mx, const real *My, const real *Mz, 
	real *Hx, real *Hy, real *Hz, 
	const real *Ms, const real *patx, const real *paty, const real *patz, int numEntries,
	int dim_x, int dim_y, int dim_z, int logical_grid_dim_y, double scale)
{
	// Thread index (inside block)
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int tz = threadIdx.z;

	// Cell index
	const int sx =  blockIdx.x                       * BLOCK_3D_SIZE_X + tx;
	const int sy = (blockIdx.y % logical_grid_dim_y) * BLOCK_3D_SIZE_Y + ty;
	const int sz = (blockIdx.y / logical_grid_dim_y) * BLOCK_3D_SIZE_Z + tz;

	// Copy into shared mem
	//__shared__ real shared[4][BLOCK_3D_SIZE_Z+2][BLOCK_3D_SIZE_Y+2][BLOCK_3D_SIZE_X+2];

	if (sx < dim_x && sy < dim_y && sz < dim_z) {
		
		// I. Prepare indices
		const int i = sx + dim_x*sy + dim_x*dim_y*sz; // center pos
		
		if (Ms[i] == 0.0) {
			Hx[i] = 0.0;
			Hy[i] = 0.0;
			Hz[i] = 0.0;
		}
		else {

			real sumx = 0.0;
			real sumy = 0.0;
			real sumz = 0.0;

			for (int pattern=0; pattern<numEntries; ++pattern) {
					
				if ( (patx[pattern] == sz) || (paty[pattern] == sz) ) {
					int interactWithLayer = -1;
					if (patx[pattern] == sz) interactWithLayer = paty[pattern];
					else interactWithLayer = patx[pattern];

					const int interact_linear = sx + dim_x*sy + dim_x*dim_y*interactWithLayer; // cell to interact with

					// calculate interlayer exchange
					if (Ms[interact_linear] != 0.0) {
						sumx += patz[pattern]*(Mx[interact_linear] / Ms[interact_linear]);
						sumy += patz[pattern]*(My[interact_linear] / Ms[interact_linear]);
						sumz += patz[pattern]*(Mz[interact_linear] / Ms[interact_linear]);
					}
				}
			}

			// Exchange field at (x,y,z)
			Hx[i] = (1.0/MU0) * (1.0/scale) * sumx / Ms[i];
			Hy[i] = (1.0/MU0) * (1.0/scale) * sumy / Ms[i];
			Hz[i] = (1.0/MU0) * (1.0/scale) * sumz / Ms[i];
		}
	}
}

//////////////////////////////////////////////////////////////////////////////
// MAIN ROUTINE THAT CALLS THE KERNELS                                      //
//////////////////////////////////////////////////////////////////////////////

template <typename real>
double interlayerExchange_multi_cuda_impl(
	int dim_x, int dim_y, int dim_z,		
	double delta_x, double delta_y, double delta_z,
	bool periodic_x, bool periodic_y, bool periodic_z,
	const Matrix &Ms,
	const VectorMatrix &intExchPat,
	int numEntries,
	const VectorMatrix &M,
	VectorMatrix &H)
{
	{
		typename Matrix_const_cuda_accessor<real>::t Ms_acc(Ms); 
		typename VectorMatrix_const_cuda_accessor<real>::t M_acc(M);
		typename VectorMatrix_const_cuda_accessor<real>::t pattern_acc(intExchPat);
		typename VectorMatrix_cuda_accessor<real>::t H_acc(H);

		const real *Mx = M_acc.ptr_x(); real *Hx = H_acc.ptr_x();
		const real *My = M_acc.ptr_y(); real *Hy = H_acc.ptr_y();
		const real *Mz = M_acc.ptr_z(); real *Hz = H_acc.ptr_z();

		const real *patx = pattern_acc.ptr_x();
		const real *paty = pattern_acc.ptr_y();
		const real *patz = pattern_acc.ptr_z();

		// Precalculate weights that are used in the kernels.
		//const real wx = static_cast<real>(2.0/MU0) / (delta_x * delta_x);
		//const real wy = static_cast<real>(2.0/MU0) / (delta_y * delta_y);
		//const real wz = static_cast<real>(2.0/MU0) / (delta_z * delta_z);


		dim3 block_dim(BLOCK_3D_SIZE_X, BLOCK_3D_SIZE_Y, BLOCK_3D_SIZE_Z);
		dim3 grid_dim(
			(dim_x + BLOCK_3D_SIZE_X-1) / BLOCK_3D_SIZE_X, 
			(dim_y + BLOCK_3D_SIZE_Y-1) / BLOCK_3D_SIZE_Y,
			(dim_z + BLOCK_3D_SIZE_Z-1) / BLOCK_3D_SIZE_Z
		);

		// Only 2-dimensional grids are supported, so ...
		const int logical_grid_dim_y = grid_dim.y;
		grid_dim.y *= grid_dim.z;
		grid_dim.z = 1;

		#define INTEXCH_MULTI_3D(bx,by,bz) if (periodic_x == bx && periodic_y == by && periodic_z == bz) kernel_interlayerExchange_multi_3d<real, bx, by, bz><<<grid_dim, block_dim>>>(Mx, My, Mz, Hx, Hy, Hz, Ms_acc.ptr(), patx, paty, patz, numEntries, dim_x, dim_y, dim_z, logical_grid_dim_y, delta_z * delta_z);
		INTEXCH_MULTI_3D(false, false, false)
		INTEXCH_MULTI_3D(false, false,  true)
		INTEXCH_MULTI_3D(false,  true, false)
		INTEXCH_MULTI_3D(false,  true,  true)
		INTEXCH_MULTI_3D( true, false, false)
		INTEXCH_MULTI_3D( true, false,  true)
		INTEXCH_MULTI_3D( true,  true, false)
		INTEXCH_MULTI_3D( true,  true,  true)
		#undef INTEXCH_MULTI_3D

		checkCudaLastError("gpu_interlayerExchange_multi(): kernel_interlayerExchange_multi_3d execution failed!");

		CUDA_THREAD_SYNCHRONIZE();
	}

	// and calculate exchange energy
	//std::cout << M.dotSum(H) <<std::endl;
	return -MU0 * M.dotSum(H) * delta_x * delta_y * delta_z;
}

double interlayerExchange_multi_cuda(
	int dim_x, int dim_y, int dim_z,		
	double delta_x, double delta_y, double delta_z,
	bool periodic_x, bool periodic_y, bool periodic_z,
	const Matrix &Ms,
	const VectorMatrix &intExchPat,
	int numEntries,
	const VectorMatrix &M,
	VectorMatrix &H,
	bool cuda64)
{

#ifdef HAVE_CUDA_64
	if (cuda64)
	return interlayerExchange_multi_cuda_impl<double>(dim_x, dim_y, dim_z, delta_x, delta_y, delta_z, periodic_x, periodic_y, periodic_z, Ms, intExchPat, numEntries, M, H);
	else
#endif
	return interlayerExchange_multi_cuda_impl<float>(dim_x, dim_y, dim_z, delta_x, delta_y, delta_z, periodic_x, periodic_y, periodic_z, Ms, intExchPat, numEntries, M, H);
}
